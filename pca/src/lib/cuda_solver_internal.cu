#include "hip/hip_runtime.h"
#include "cuda_solver_internal.hpp"

// cuSOLVER handle and CUDA stream are initialized on construction
CUDASolverInternal::CUDASolverInternal() {
    hipsolverStatus_t cusolver_status;
    hipError_t cuda_error;
    hipblasStatus_t cublas_status;
    hipsparseStatus_t cusparse_status;
    
    cusolver_status = hipsolverDnCreate(&cusolver_handle);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cuda_error = hipStreamCreate(&hip_stream);
    assert(cuda_error == hipSuccess);

    cublas_status = hipblasCreate(&cublas_handle);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    cusparse_status = hipsparseCreate(&cusparse_handle);
    assert(cusparse_status == HIPSPARSE_STATUS_SUCCESS);
}

// cuSOLVER handle and CUDA stream are destoryed on descruction
CUDASolverInternal::~CUDASolverInternal() {
    hipsolverStatus_t cusolver_status;
    hipError_t cuda_error;
    hipblasStatus_t cublas_status;
    hipsparseStatus_t cusparse_status;

    cusolver_status = hipsolverDnDestroy(cusolver_handle);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cuda_error = hipStreamDestroy(hip_stream);
    assert(cuda_error == hipSuccess);

    cublas_status = hipblasDestroy(cublas_handle);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    cusparse_status = hipsparseDestroy(cusparse_handle);
    assert(cusparse_status == HIPSPARSE_STATUS_SUCCESS);
}

// Prints information of the i-th CUDA device
void CUDASolverInternal::printDeviceInfo(int i) {
    int device_count;
    hipGetDeviceCount(&device_count);

    assert(i >= 0 && i < device_count);

    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, i);

    std::cout << "Name: " << device_prop.name << std::endl;
    std::cout << "Compute capability: " << device_prop.major << "." << device_prop.minor
        << " (should be 1.3 or above to support double precision)" << std::endl;
    std::cout << "Clock rate: " << device_prop.clockRate << " kHz" << std::endl;
    std::cout << "Device copy overlap: ";
    if(device_prop.deviceOverlap) std::cout << "Enabled" << std::endl;
    else std::cout << "Disabled" << std::endl;
    std::cout << "Kernel execution timeout: ";
    if(device_prop.kernelExecTimeoutEnabled) std::cout << "Enabled" << std::endl;
    else std::cout << "Disabled" << std::endl;
    std::cout << "Total global mem: " << device_prop.totalGlobalMem << " B" << std::endl;
    std::cout << "Total constant mem: " << device_prop.totalConstMem << " B" << std::endl;
    std::cout << "Multiprocessor count: " << device_prop.multiProcessorCount << std::endl;
    std::cout << "Shared mem per mp: " << device_prop.sharedMemPerBlock << " B" << std::endl;
    std::cout << "Registers per mp: " << device_prop.regsPerMultiprocessor << std::endl;
    std::cout << "Threads in warp: " << device_prop.warpSize << std::endl;
    std::cout << "Max threads per block: " << device_prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max thread dimensions: " << device_prop.maxThreadsDim[0] << " " << device_prop.maxThreadsDim[1] << " " << device_prop.maxThreadsDim[2] << std::endl;
    std::cout << "Max grid dimensions: " << device_prop.maxGridSize[0] << " " << device_prop.maxGridSize[1] << " " << device_prop.maxGridSize[2] << std::endl;

    return;
}

// Solves Ax = b via Cholesky decomposition
// Assumes that A is an n x n positive definite matrix
// Implemented with cuSOLVER library
void CUDASolverInternal::solveCholesky(int n, const double *A, const double *b, double *x) {
    hipsolverStatus_t cusolver_status;
    hipError_t cuda_error;

    // Assuming that the leading dimension is equal to n
    int lda = n;

    // Preparing device memories
    double *d_A = NULL;
    double *d_b = NULL;
    double *d_x = NULL;

    cuda_error = hipMalloc((void **)&d_A, sizeof(double)*lda*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_b, sizeof(double)*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_x, sizeof(double)*n);
    assert(cuda_error == hipSuccess);

    // Copying A and b from host to device
    cuda_error = hipMemcpy(d_A, A, sizeof(double)*lda*n, hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMemcpy(d_b, b, sizeof(double)*n, hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);

    // Preparing cuSOLVER-related memories
    int buffer_size = 0;
    double *d_buffer = NULL;
    int *d_info = NULL;
    int info = 0;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    
    // Calculating the buffer size needed
    cusolver_status = hipsolverDnDpotrf_bufferSize(cusolver_handle, uplo, n, (double*)A, n, &buffer_size);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cuda_error = hipMalloc(&d_info, sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc(&d_buffer, sizeof(double) * buffer_size);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMemset(d_info, 0, sizeof(int));
    assert(cuda_error == hipSuccess);

    // Computes Cholesky decomposition
    cusolver_status = hipsolverDnDpotrf(cusolver_handle, uplo, n, d_A, lda, d_buffer, buffer_size, d_info);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cuda_error = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(cuda_error == hipSuccess);
    assert(info == 0);

    cuda_error = hipMemcpy(d_x, d_b, sizeof(double)*n, hipMemcpyDeviceToDevice);
    assert(cuda_error == hipSuccess);

    // Solving Ax = b
    cusolver_status = hipsolverDnDpotrs(cusolver_handle, uplo, n, 1, d_A, lda, d_x, n, d_info);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cuda_error = hipDeviceSynchronize();
    assert(cuda_error == hipSuccess);

    // Copying the result x from device to host
    cuda_error = hipMemcpy(x, d_x, sizeof(double)*n, hipMemcpyDeviceToHost);
    assert(cuda_error == hipSuccess);

    // Freeing the device memories
    cuda_error = hipFree(d_info);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_buffer);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_A);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_b);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_x);
    assert(cuda_error == hipSuccess);
    return;
}

// Solves Ax = b via LU decomposition
// Assumes that A is a square matrix
// Implemented with cuSOLVER library
void CUDASolverInternal::solveLU(int n, const double *A, const double *b, double *x) {
    hipsolverStatus_t cusolver_status;
    hipError_t cuda_error;

    // Assuming that the leading dimension is equal to n
    int lda = n;

    // Preparing device memories
    double *d_A = NULL;
    double *d_b = NULL;
    double *d_x = NULL;

    cuda_error = hipMalloc((void **)&d_A, sizeof(double)*lda*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_b, sizeof(double)*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_x, sizeof(double)*n);
    assert(cuda_error == hipSuccess);

    // Copying A and b from host to device
    cuda_error = hipMemcpy(d_A, A, sizeof(double)*lda*n, hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMemcpy(d_b, b, sizeof(double)*n, hipMemcpyHostToDevice);
    assert(cuda_error == hipSuccess);

    // Preparing cuSOLVER-related memories
    int buffer_size = 0;
    double *d_buffer = NULL;
    int *d_info = NULL;
    int info = 0;
    int *d_ipiv = NULL;  // Pivoting sequence
    
    // Calculating the buffer size needed
    cusolver_status = hipsolverDnDgetrf_bufferSize(cusolver_handle, n, n, (double*)A, lda, &buffer_size);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cuda_error = hipMalloc(&d_info, sizeof(int));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc(&d_buffer, sizeof(double) * buffer_size);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc(&d_ipiv, sizeof(int)*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMemset(d_info, 0, sizeof(int));
    assert(cuda_error == hipSuccess);

    // Computes LU decomposition
    cusolver_status = hipsolverDnDgetrf(cusolver_handle, n, n, d_A, lda, d_buffer, d_ipiv, d_info);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cuda_error = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(cuda_error == hipSuccess);
    assert(info == 0);

    cuda_error = hipMemcpy(d_x, d_b, sizeof(double)*n, hipMemcpyDeviceToDevice);
    assert(cuda_error == hipSuccess);

    // Solving Ax = b
    cusolver_status = hipsolverDnDgetrs(cusolver_handle, HIPBLAS_OP_N, n, 1, d_A, lda, d_ipiv, d_x, n, d_info);
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cuda_error = hipDeviceSynchronize();
    assert(cuda_error == hipSuccess);

    // Copying the result x from device to host
    cuda_error = hipMemcpy(x, d_x, sizeof(double)*n, hipMemcpyDeviceToHost);
    assert(cuda_error == hipSuccess);

    // Freeing the device memories
    cuda_error = hipFree(d_info);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_buffer);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_A);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_b);
    assert(cuda_error == hipSuccess);
    cuda_error = hipFree(d_x);
    assert(cuda_error == hipSuccess);
    return;
}

// Solves Ax = b via conjugate gradient
// Assumes that A is a n x n symmetric positive definite sparse matrix
// Implemented with cuSPARSE library
void CUDASolverInternal::solveConjugateGradient(int n, int nz, const int *I, const int *J, const double *val, const double *rhs, double *x) {
    hipsparseStatus_t cusparse_status;
    hipblasStatus_t cublas_status;
    hipError_t cuda_error;
    
    hipsparseMatDescr_t descr = 0;
    cusparse_status = hipsparseCreateMatDescr(&descr);
    assert(cusparse_status == HIPSPARSE_STATUS_SUCCESS);

    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_SYMMETRIC);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int *d_col = NULL;
    int *d_row = NULL;
    double *d_val = NULL;
    double *d_x = NULL;
    double *d_r = NULL;
    double *d_p = NULL;
    double *d_Ax = NULL;

    cuda_error = hipMalloc((void **)&d_col, sizeof(int)*nz);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_row, sizeof(int)*(n+1));
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_val, sizeof(double)*nz);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_x, sizeof(double)*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_r, sizeof(double)*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_p, sizeof(double)*n);
    assert(cuda_error == hipSuccess);
    cuda_error = hipMalloc((void **)&d_Ax, sizeof(double)*n);
    assert(cuda_error == hipSuccess);

    hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row, I, (n+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_val, val, nz*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_r, rhs, n*sizeof(double), hipMemcpyHostToDevice);

    double alpha = 1.0, alpham1 = -1.0, beta = 0.0, r0 = 0.0, r1;

    hipsparseDcsrmv(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);

    hipblasDaxpy(cublas_handle, n, &alpham1, d_Ax, 1, d_r, 1);
    cublas_status = hipblasDdot(cublas_handle, n, d_r, 1, d_r, 1, &r1);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    int k = 1;
    const double tol = 0.00001;
    const int max_iter = 10000;
    double a, b, na, dot;

    while (r1 > tol*tol && k <= max_iter) {
        if (k > 1) {
            b = r1 / r0;
            cublas_status = hipblasDscal(cublas_handle, n, &b, d_p, 1);
            assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
            cublas_status = hipblasDaxpy(cublas_handle, n, &alpha, d_r, 1, d_p, 1);
            assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        } else {
            cublas_status = hipblasDcopy(cublas_handle, n, d_r, 1, d_p, 1);
            assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        }

        hipsparseDcsrmv(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax);
        cublas_status = hipblasDdot(cublas_handle, n, d_p, 1, d_Ax, 1, &dot);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        a = r1 / dot;

        cublas_status = hipblasDaxpy(cublas_handle, n, &a, d_p, 1, d_x, 1);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        na = -a;
        cublas_status = hipblasDaxpy(cublas_handle, n, &na, d_Ax, 1, d_r, 1);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

        r0 = r1;
        cublas_status = hipblasDdot(cublas_handle, n, d_r, 1, d_r, 1, &r1);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        hipDeviceSynchronize();
        //printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    hipMemcpy(x, d_x, n*sizeof(double), hipMemcpyDeviceToHost);

    double rsum, diff, err = 0.0;

    for (int i = 0; i < n; i++) {
        rsum = 0.0;

        for (int j = I[i]; j < I[i+1]; j++) {
            rsum += val[j]*x[J[j]];
        }

        diff = fabs(rsum - rhs[i]);

        if (diff > err) {
            err = diff;
        }
    }

    //printf("Test Summary:  Error amount = %lf\n", err);

    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

    return;
}