#include "hip/hip_runtime.h"
// Copyright (c) 2009, Graham Markall and Tristan Perryman
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//  * Redistributions of source code must retain the above copyright notice, this list
//    of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright notice, this
//    list of conditions and the following disclaimer in the documentation and/or 
//    other materials provided with the distribution.
//  * Neither the name of Imperial College London nor the names of its contributors
//    may be used to endorse or promote products derived from this software without 
//    specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY 
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES 
// OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT 
// SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, 
// INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED 
// TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR 
// BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN 
// CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN 
// ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH
// DAMAGE.

// gpu_solve.cu
// Implements a Jacobi preconditioned sparse conjugate gradient solver on the GPU.
// Based on a solver originally produced by Tristan Perryman.
// Jacobi preconditioning added and code refactored by Graham Markall.

#include<stdio.h>
#include<sys/time.h>

// Texture references for CSR matrix 
texture<int,1> tex_colm;
texture<int2,1> tex_val;

// Scratchpad used by vector dot product for reduction
double* scratchpad;

// Kernel block and grid parameters - threads in a block and blocks in a grid
#define NUM_THREADS 128
#define NUM_BLOCKS 16

// Macros to simplify kernels 
#define THREAD_ID threadIdx.x+blockIdx.x*blockDim.x
#define THREAD_COUNT gridDim.x*blockDim.x

// Solver parameters - relative tolerance and maximum iterations
#define epsilon 1e-6
#define IMAX 10000

// For timing solver
double utime () {
  struct timeval tv;

  gettimeofday (&tv, NULL);

  return (tv.tv_sec + double (tv.tv_usec) * 1e-6);
}

// Creates a diagonal matrix stored in a vector pcmat, from the CSR matrix findrm, colm, val.
// n is the matrix size.
__global__ void create_jac(int n, int* findrm, int* colm, double* val, double* pcmat)
{
  for(int i=THREAD_ID; i<n; i+=THREAD_COUNT) 
    for(int k=findrm[i]-1; k<findrm[i+1]-1; k++) 
      if(colm[k]==i+1)
        pcmat[i] = 1.0/val[k];
}

// Multiplies diagonal matrix mat stored as a vector by the vector src, storing result in dest.
// n is the vector length.
__global__ void diag_spmv(int n, double *mat, double *src, double *dest) 
{
  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    dest[i] = mat[i]*src[i];
}

// Sets the length-n vector vec to the zero vector.
__global__ void veczero(int n, double* vec) 
{
  for(int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    vec[i] = 0;
}

// Allows fetching double values from texture memory, which only supports integers
static __device__ double fetch_double(texture<int2,1> val, int elem)
{
  int2 v = tex1Dfetch(val, elem);
  return __hiloint2double(v.y, v.x);
}

// Multiplies the CSR matrix in findrm, tex_colm, tex_val by src and stores the
// result in dest. n is the matrix size/vector length.
__global__ void csr_spmv(int n, double* src, double* dest, int *findrm)
{
  for (int row=THREAD_ID; row<n; row+=THREAD_COUNT) {
    dest[row] = 0;
    int a=findrm[row];
    int b=findrm[row+1];
    for (int k=a;k<b;k++)
      dest[row] += fetch_double(tex_val,k-1)*src[tex1Dfetch(tex_colm,k-1)-1];
  }
}

// Computes the dot product of length-n vectors vec1 and vec2. This is reduced in tmp into a
// single value per thread block. The reduced value is stored in the array partial.
__global__ void vecdot_partial(int n, double* vec1, double* vec2, double* partial)
{ 
  __shared__ double tmp[NUM_THREADS];
  tmp[threadIdx.x] = 0;

  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    tmp[threadIdx.x] += vec1[i]*vec2[i];
  
  for (int i=blockDim.x/2;i>=1;i = i/2) {
    __syncthreads();
    if (threadIdx.x < i) 
      tmp[threadIdx.x] += tmp[i + threadIdx.x]; 
  }
  
  if (threadIdx.x == 0) 
    partial[blockIdx.x] = tmp[0];
}

// Reduces the output of the vecdot_partial kernel to a single value. The result is stored in result.
__global__ void vecdot_reduce(double* partial, double* result)
{
  __shared__ double tmp[NUM_BLOCKS];
  
  if (threadIdx.x < NUM_BLOCKS) 
    tmp[threadIdx.x] = partial[threadIdx.x]; 
  else 
    tmp[threadIdx.x] = 0;
  
  for (int i=blockDim.x/2;i>=1;i = i/2) {
    __syncthreads();
    if (threadIdx.x < i) 
      tmp[threadIdx.x] += tmp[i + threadIdx.x]; 
  }
  
  if (threadIdx.x == 0) 
    *result = tmp[0];
}

// Divides num by den and stores the result in result. This is very wasteful of the GPU.
__global__ void scalardiv(double* num, double* den, double* result) 
{
  if(threadIdx.x==0 && blockIdx.x==0)
    *result = (*num)/(*den);
}

// Computes r= a*x+y for n-length vectors x and y, and scalar a.
__global__ void axpy(int n, double* a, double* x, double* y, double* r) 
{
  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    r[i] = y[i] + (*a)*x[i];
}

// Computes y= y-a*x for n-length vectors x and y, and scalar a.
__global__ void ymax(int n, double* a, double* x, double* y) 
{
  for (int i=THREAD_ID; i<n; i+=THREAD_COUNT)
    y[i] = y[i] - (*a)*x[i];
}

// Convenient function for performing a vector dot product and reduce all in one go.
void vecdot(int n, double* vec1, double* vec2, double* result) 
{ 
  dim3 BlockDim(NUM_THREADS);
  dim3 GridDim(NUM_BLOCKS);
  
  vecdot_partial<<<GridDim,BlockDim>>>(n, vec1, vec2, scratchpad);
  vecdot_reduce<<<1,NUM_BLOCKS>>>(scratchpad, result);
}

// Sets dest=src for scalars on the GPU.
void scalarassign(double* dest, double* src)
{
  hipMemcpy(dest, src, sizeof(double), hipMemcpyDeviceToDevice); 
}

// Sets dest=src for n-length vectors on the GPU.
void vecassign(double *dest, double *src, int n) 
{
  hipMemcpy(dest, src, sizeof(double)*n, hipMemcpyDeviceToDevice);
}


// Main solver entry point. Solves Ax = b for x.
// CSR matrix (A) stores the row and column pointer information like Fortran indices (i.e. begin at 1)
// findrm_p     : matrix row pointer
// colm_p       : matrix column pointer
// matrix_val_p : matrix values
// size*        : size of each vector
// b_p          : pointer to RHS vector
// x_p          : solution (x) is returned here
extern "C" int gpucg_solve_(int* findrm_p, int *size_findrm, int* colm_p, int* size_colm, double* matrix_val_p, int *matrix_val_size, 
                             double* b_p, int* rhs_val_size, double *x_p)
{
  // CSR Matrix on the GPU
  int *k_findrm, *k_colm;
  double *k_val;
  // Vectors on the GPU
  double *k_b, *k_x, *k_r, *k_d, *k_q, *k_s;
  // Diagonal matrix on the GPU (stored as a vector)
  double* k_jac;
  // Scalars on the GPU
  double  *k_alpha, *k_snew, *k_beta, *k_sold, *k_s0;
  
  // Scalars on the host
  double t, s0, snew;
  int iterations = 0;

  // Begin timing
  t = -utime ();
  
  // Allocate space on the GPU for the CSR matrix and RHS vector, and copy from host to GPU
  hipMalloc((void**)&k_findrm, sizeof(int)*(*size_findrm));
  hipMemcpy(k_findrm, findrm_p, sizeof(int)*(*size_findrm), hipMemcpyHostToDevice);
  hipMalloc((void**)&k_colm, sizeof(int)*(*size_colm));
  hipMemcpy(k_colm, colm_p, sizeof(int)*(*size_colm), hipMemcpyHostToDevice);
  hipBindTexture(NULL, tex_colm, k_colm, sizeof(int)*(*size_colm));
  hipMalloc((void**)&k_val, sizeof(double)*(*matrix_val_size));
  hipMemcpy(k_val, matrix_val_p, sizeof(double)*(*matrix_val_size), hipMemcpyHostToDevice);
  hipMalloc((void**)&k_b, sizeof(double)*(*rhs_val_size));
  hipMemcpy(k_b, b_p, sizeof(double)*(*rhs_val_size), hipMemcpyHostToDevice);

  // Allocate space for vectors on the GPU
  hipMalloc((void**)&k_x, sizeof(double)*(*rhs_val_size));
  hipMalloc((void**)&k_r, sizeof(double)*(*rhs_val_size));
  hipMalloc((void**)&k_d, sizeof(double)*(*rhs_val_size));
  hipMalloc((void**)&k_q, sizeof(double)*(*rhs_val_size));
  hipMalloc((void**)&k_s, sizeof(double)*(*rhs_val_size));
  hipMalloc((void**)&k_jac, sizeof(double)*(*rhs_val_size));
  hipMalloc((void**)&k_alpha, sizeof(double));
  hipMalloc((void**)&scratchpad, sizeof(double)*NUM_BLOCKS);
  hipMalloc((void**)&k_snew, sizeof(double)*NUM_BLOCKS);
  hipMalloc((void**)&k_sold, sizeof(double));
  hipMalloc((void**)&k_beta, sizeof(double));
  hipMalloc((void**)&k_s0, sizeof(double));

  // Dimensions of blocks and grid on the GPU
  dim3 BlockDim(NUM_THREADS);
  dim3 GridDim(NUM_BLOCKS);

  // Create diagonal preconditioning matrix (J = 1/diag(M)) 
  create_jac<<<1,BlockDim>>>(*rhs_val_size, k_findrm, k_colm, k_val, k_jac);
  
  // Bind the matrix to the texture cache - this was not done earlier as we modified the matrix
  hipBindTexture(NULL, tex_val, k_val, sizeof(double)*(*matrix_val_size)); 
  
  // Initialise result vector (x=0)
  veczero<<<1,BlockDim>>>(*rhs_val_size, k_x);

  // r=b-Ax (r=b since x=0), and d=M^(-1)r
  hipMemcpy(k_r, k_b, sizeof(double)*(*rhs_val_size), hipMemcpyDeviceToDevice);
  diag_spmv<<<1,BlockDim>>>(*rhs_val_size, k_jac, k_r, k_d);

  // s0 = r.d
  vecdot(*rhs_val_size, k_r, k_d, k_s0);
  // snew = s0
  scalarassign(k_snew, k_s0);
  printf("CUDA error 1: %s \n", hipGetErrorString(hipGetLastError()));
  // Copy snew and s0 back to host so that host can evaluate stopping condition
  hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
  printf("CUDA error 2: %s \n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(&s0, k_s0, sizeof(double), hipMemcpyDeviceToHost);
  printf("CUDA error 3: %s \n", hipGetErrorString(hipGetLastError()));

  // While i < imax and snew > epsilon^2*s0
  while (iterations < IMAX && snew > epsilon*epsilon*s0)
  {
    // q = Ad
    csr_spmv<<<GridDim,BlockDim>>>(*rhs_val_size, k_d, k_q, k_findrm);
    // alpha = snew/(d.q)
    vecdot(*rhs_val_size, k_d, k_q, k_alpha);
    scalardiv<<<1,1>>>(k_snew, k_alpha, k_alpha);
    // x = x + alpha*d
    axpy<<<GridDim,BlockDim>>>(*rhs_val_size, k_alpha, k_d, k_x, k_x);
    // r = r - alpha*q
    ymax<<<GridDim,BlockDim>>>(*rhs_val_size, k_alpha, k_q, k_r);
    // s = M^(-1)r
    diag_spmv<<<GridDim,BlockDim>>>(*rhs_val_size, k_jac, k_r, k_s);
    // sold = snew
    scalarassign(k_sold, k_snew);
    // snew = r.s
    vecdot(*rhs_val_size, k_r, k_s, k_snew);
    // beta = snew/sold
    scalardiv<<<1,1>>>(k_snew, k_sold, k_beta);
    // d = s + beta*d
    axpy<<<GridDim,BlockDim>>>(*rhs_val_size, k_beta, k_d, k_s, k_d);
    // Copy back snew so the host can evaluate the stopping condition
    hipMemcpy(&snew, k_snew, sizeof(double), hipMemcpyDeviceToHost);
    // i = i+1
    iterations++;
  }
  
  // Copy result vector back from GPU
  hipMemcpy(x_p, k_x, sizeof(double)*(*rhs_val_size), hipMemcpyDeviceToHost);
  
  // Clean up
  hipUnbindTexture(tex_colm);
  hipUnbindTexture(tex_val);
  hipFree(k_findrm);
  hipFree(k_colm);
  hipFree(k_val);
  hipFree(k_b);
  hipFree(k_x);
  hipFree(k_r);
  hipFree(k_d);
  hipFree(k_q);
  hipFree(k_s);
  hipFree(k_jac);
  hipFree(k_alpha);
  hipFree(k_snew);
  hipFree(k_sold);
  hipFree(k_beta);
  hipFree(k_s0);
  hipFree(scratchpad);

  // End timing - call hipDeviceSynchronize so we know all computation is finished before we stop the clock.
  hipDeviceSynchronize();
  t += utime ();

  // Interesting information
  printf("Iterations: %d \n", iterations);
  printf("Solve time: %f seconds\n", t);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
  {
    printf("CUDA error is: %s \n", hipGetErrorString(error));
    return EXIT_FAILURE;
  }
  return EXIT_SUCCESS;
}

